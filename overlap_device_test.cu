#include <stdio.h>
#include "book.h"

int main(void)
{
    int whichDevice;
    hipDeviceProp_t prop;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap)
    {
        printf("Device do not supports device overlap\n");
    }
    else
    {
        printf("Passed\n");
    }
}

