#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#define N 10
#define DIM 5
#define OUTPUT_DIM 3

texture<float, 2> tex;

void printMatrix(float* p, int row, int column)
{
    for (int i = 0; i < row; i++)
    {
        printf("[ ");
        for (int j = 0; j < column; j++)
        {
            if (j != column - 1)
            {
                printf("%.3f, ", *(p + i * column + j));
            }
            else
            {
                printf("%.3f ]\n", *(p + i * column + j));
            }
        }
    }
}

__global__ void AveragePooling2D(float* p_result)
{
    // pos为输出中的位置
    int pos = threadIdx.y * blockDim.x + threadIdx.x;
    // 计算对应输入区域的中心位置
    int x = threadIdx.x + 1;
    int y = threadIdx.y + 1;
    float result = 0;
    int start = OUTPUT_DIM / 2 * (-1); 
    int stop = OUTPUT_DIM / 2;
    for (int i = start; i <= stop; i++)
    {
        for (int j = start; j <= stop; j++)
        {
            result += tex2D(tex, x + i, y + j);
        }
    }
    result /= (OUTPUT_DIM * OUTPUT_DIM);
    *(p_result + pos) = result;
}

int main(void)
{
    float* host_p;
    float* host_r;
    float* dev_p;
    float* dev_r;
    int error = 0;
    // 创建主机上的页锁定内存 用于异步的和GPU之间进行数据拷贝
    error = hipHostAlloc((void**)&host_p, N * DIM * DIM * sizeof(float), hipHostMallocDefault);
    printf("Error code hipHostAlloc: %d\n", error);
    for (int k = 0; k < N; k++)
    {
        float* tmp = host_p + k * DIM * DIM;
        for (int i = 0; i < DIM; i++)
        {
            for (int j =0; j < DIM; j++)
            {
                *(tmp + i * DIM + j) = 100.0 * k;
            }
        }
    }
    for (int k = 0; k < N; k++)
    {
        printMatrix(host_p + k * DIM * DIM, DIM, DIM);
    }
    error = hipHostAlloc((void**)&host_r, N * OUTPUT_DIM * OUTPUT_DIM * sizeof(float), hipHostMallocDefault);
    printf("Error code hipHostAlloc: %d\n", error);
    // 创建GPU上的缓存空间
    size_t pitch;
    error = hipMallocPitch((void**)&dev_p, &pitch, DIM * sizeof(float), DIM);
    printf("Error code hipMallocPitch: %d\n", error);
    error = hipMalloc((void**)&dev_r, OUTPUT_DIM * OUTPUT_DIM * sizeof(float));
    printf("Error code hipMalloc: %d\n", error);
    // 对于输入 将其绑定到二维纹理内存上
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    error = hipBindTexture2D(NULL, tex, dev_p, desc, DIM, DIM, pitch);
    printf("Error code hipBindTexture2D: %d\n", error);
    // 创建cuda流
    // hipStream_t stream;
    // hipStreamCreate(&stream);

    // 创建cuda事件 用于计时
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // 迭代计算N个输入
    for (int i = 0; i < N; i++)
    {
        // 通过同步方式将输入数据copy至GPU
        error = hipMemcpy2D(dev_p, pitch, host_p + i * DIM * DIM, DIM * sizeof(float), DIM * sizeof(float), DIM, hipMemcpyHostToDevice);
        printf("Error code hipMemcpy2D: %d\n", error);
        // 调用核函数
        dim3 threads(OUTPUT_DIM, OUTPUT_DIM);
        AveragePooling2D<<<1, threads>>>(dev_r);
        // 通过异步方式将输出数据copy至CPU
        error = hipMemcpy(host_r + i * OUTPUT_DIM * OUTPUT_DIM, dev_r, OUTPUT_DIM * OUTPUT_DIM * sizeof(float), hipMemcpyDeviceToHost);
        printf("Error code hipMemcpy: %d\n", error);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf( "Time Cost:  %3.1f ms\n", elapsedTime );
    hipUnbindTexture(tex);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    // hipStreamDestroy(stream);
    hipFree(dev_p);
    hipFree(dev_r);
    for (int k = 0; k < N; k++)
    {
        printMatrix(host_r + k * OUTPUT_DIM * OUTPUT_DIM, OUTPUT_DIM, OUTPUT_DIM);
    }
    hipHostFree(host_p);
    hipHostFree(host_r);
    return 0;
}

