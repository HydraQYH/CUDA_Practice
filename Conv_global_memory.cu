#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#define FEATURE_DIM 38
#define FILTER_DIM 7
#define STRIDE 1
#define PADDING 0

void printMatrix(float* p, int row, int column)
{
    for (int i = 0; i < row; i++)
    {
        printf("[ ");
        for (int j = 0; j < column; j++)
        {
            if (j != column - 1)
            {
                printf("%.3f, ", *(p + i * column + j));
            }
            else
            {
                printf("%.3f ]\n", *(p + i * column + j));
            }
        }
    }
}

__global__ void conv(float* p_input, float* p_filter, float* p_output)
{
    // 当前线程要计算输出中(threadIdx.x, threadIdx.y)的值
    int output_offset = threadIdx.y + threadIdx.x * blockDim.y;
    // 获取当前输出位置所对应的输入区域中心
    int input_x_offset = FILTER_DIM/ 2 + STRIDE * threadIdx.x;
    int input_y_offset = FILTER_DIM / 2 + STRIDE * threadIdx.y;
    // 区域左上角
    int input_x_offset_left_up = input_x_offset - FILTER_DIM / 2;
    int input_y_offset_left_up = input_y_offset - FILTER_DIM / 2;

    float result = 0;
    for (int i = 0; i < FILTER_DIM; i++)
    {
        for (int j = 0; j < FILTER_DIM; j++)
        {
            int x = input_x_offset_left_up + i;
            int y = input_y_offset_left_up + j;
            int offset = x * (FEATURE_DIM + 2 * PADDING) + y;
            result += p_input[offset] * p_filter[i * FILTER_DIM + j];
        }
    }
    p_output[output_offset] = result;
}

int main(void)
{
    // 创建输入图像
    float input[FEATURE_DIM + 2 * PADDING][FEATURE_DIM + 2 * PADDING];
    for (int i = 0; i < FEATURE_DIM + 2 * PADDING; i++)
    {
        for (int j = 0; j < FEATURE_DIM + 2 * PADDING; j++)
        {
            // input[i][j] = i * (FEATURE_DIM + 2 * PADDING) + 1 + j;
            input[i][j] = 1;
        }
    }
    printf("Input Feature Map:\n");
    printMatrix((float*)input, FEATURE_DIM + 2 * PADDING, FEATURE_DIM + 2 * PADDING);
    // 创建卷积核
    float filter[FILTER_DIM][FILTER_DIM];
    for (int i = 0; i < FILTER_DIM; i++)
    {
        for (int j = 0; j < FILTER_DIM; j++)
        {
            // filter[i][j] = i * FILTER_DIM + 1 + j;
            filter[i][j] = 1;
        }
    }
    printf("Filter:\n");
    printMatrix((float*)filter, FILTER_DIM, FILTER_DIM);
    // 创建输出
    int output_dim = (FEATURE_DIM + 2 *PADDING - FILTER_DIM) / STRIDE + 1;
    float* p_output = (float*)malloc(output_dim * output_dim * sizeof(float));

    float* input_dev;
    float* filter_dev;
    float* output_dev;

    // 在GPU上为输入输出以及卷积核分配空间
    hipMalloc((void**)&input_dev, (FEATURE_DIM + 2 * PADDING) * (FEATURE_DIM + 2 * PADDING) * sizeof(float));
    hipMalloc((void**)&filter_dev, (FILTER_DIM) * (FILTER_DIM) * sizeof(float));
    hipMalloc((void**)&output_dev, output_dim * output_dim * sizeof(float));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    hipMemcpy(input_dev, input, (FEATURE_DIM + 2 * PADDING) * (FEATURE_DIM + 2 * PADDING) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(filter_dev, filter, FILTER_DIM * FILTER_DIM * sizeof(float), hipMemcpyHostToDevice);
    // 只使用一个线程块
    dim3 threads(output_dim, output_dim);
    conv<<<1, threads>>>(input_dev, filter_dev, output_dev);
    hipMemcpy(p_output, output_dev, output_dim * output_dim * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    hipFree(input_dev);
    hipFree(filter_dev);
    hipFree(output_dev);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Output Feature map:\n");
    printMatrix((float*)p_output, output_dim, output_dim);
    free(p_output);
    printf( "Time Cost:  %3.1f ms\n", elapsedTime );
    return 0;
}

